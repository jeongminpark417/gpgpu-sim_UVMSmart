/**
 * 2DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#define NI 1024
#define NJ 1024

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

__global__ void Test_kernel(DATA_TYPE *A, DATA_TYPE *B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	B[i] = A[i];	
}


void Test(DATA_TYPE* A, DATA_TYPE* B)
{
	
	Test_kernel<<<10, 64>>>(A, B);

	hipDeviceSynchronize();
}


int main(int argc, char *argv[])
{
	DATA_TYPE* A;
	DATA_TYPE* B;  

	hipMallocManaged( &A, NI*NJ*sizeof(DATA_TYPE) );
	hipMallocManaged( &B, NI*NJ*sizeof(DATA_TYPE) );

	Test(A, B);
	

	hipFree(A);
	hipFree(B);
	
	return 0;
}

