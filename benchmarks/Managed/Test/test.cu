/**
 * 2DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#define NI 1024
#define NJ 1024

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

#define clock_value_t  long long;

__device__ void c_sleep(long long sleep_cycles)
{
    long long start = clock64();
    long long cycles_elapsed;
    do { cycles_elapsed = clock64() - start; } 
    while (cycles_elapsed < sleep_cycles);
}

__global__ void Test_kernel(DATA_TYPE *A, DATA_TYPE *B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int a1 = A[i];
	int a2 = A[i+1];
	
//	if(threadIdx.x != 0) return;

	if(a1 < a2){
		for(int k = a1; a1 < a2; a1++){
			a2 = (a1 | a2);
		}
	}
	else{
		int a3 = a1;
		a1 = a2;
		a2 = a3;
		for(int k = a1; a1 < a2; a1++){ 
			a2 = (a1 | a2); 
		}
	}
	int temp = (a1 + A[i]) + (a2 + A[i + 1]);	
	int x = 0;
	for(int k = 0; k < 100; k++){
		if ((temp)  == 0) x++;	
	}	
	B[i] =temp + x;


//c_sleep(10000);	

	
}


void Test(DATA_TYPE* A, DATA_TYPE* B)
{
	
	Test_kernel<<<10, 32>>>(A, B);

	hipDeviceSynchronize();
}


int main(int argc, char *argv[])
{
	DATA_TYPE* A;
	DATA_TYPE* B;  

	hipMallocManaged( &A, 2*NI*NJ*sizeof(DATA_TYPE) );
	hipMallocManaged( &B, 2*NI*NJ*sizeof(DATA_TYPE) );

	Test(A, B);
	

	hipFree(A);
	hipFree(B);
	
	return 0;
}

