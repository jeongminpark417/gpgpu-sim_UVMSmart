/**
 * 2DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#define NI 2048
#define NJ 2048

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef int DATA_TYPE;

#define clock_value_t  long long;

__device__ void c_sleep(long long sleep_cycles)
{
    long long start = clock64();
    long long cycles_elapsed;
    do { cycles_elapsed = clock64() - start; } 
    while (cycles_elapsed < sleep_cycles);
}

__global__ void Test_kernel(DATA_TYPE *A, DATA_TYPE *B, int* C)
{

	__shared__ int s_temp;

	long bidx = blockIdx.x;
	for(int itr = 0; itr < 1; itr++){
		int temp2 = 0;
			for(int j = 0; j < 5; j++){
			temp2 += B[bidx*1024*512 + threadIdx.x + j * 1024*514+itr];
			}
		C[bidx * 64 + threadIdx.x] = temp2 +  C[bidx * 64 + threadIdx.x] ;
		__syncthreads();	

		if(threadIdx.x == 0){
			int temp = 0;
			for(int j = 0; j < 5;j++){
				temp += A[(bidx + itr % 3) * 1024 * 512 * j];
			}
			s_temp = temp;
		}
		__syncthreads();
		//reguar
		 temp2 = 0;
			for(int j = 0; j < 5; j++){
			temp2 += B[bidx*1024*512 + threadIdx.x + j * 1024*514+itr];
			}
		C[bidx * 64 + threadIdx.x + 1] = temp2 + C[bidx * 64 + threadIdx.x + 1];
		__syncthreads();	
	}	
}


void Test(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C)
{
	
	Test_kernel<<<1, 128>>>(A, B, C);

	hipDeviceSynchronize();
}


int main(int argc, char *argv[])
{
	DATA_TYPE* A;
	DATA_TYPE* B;  
	DATA_TYPE* C;
	
	hipMallocManaged( &A, 2*NI*NJ*sizeof(DATA_TYPE) );
	hipMallocManaged( &B, 2*NI*NJ*sizeof(DATA_TYPE) );
	hipMallocManaged( &C, 2*NI*NJ*sizeof(DATA_TYPE) );


	Test(A, B,C);
	

	hipFree(A);
	hipFree(B);
	
	return 0;
}

